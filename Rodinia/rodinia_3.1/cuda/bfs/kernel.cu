#include "hip/hip_runtime.h"
/*********************************************************************************
Implementing Breadth first search on CUDA using algorithm given in HiPC'07
  paper "Accelerating Large Graph Algorithms on the GPU using CUDA"

Copyright (c) 2008 International Institute of Information Technology - Hyderabad. 
All rights reserved.
  
Permission to use, copy, modify and distribute this software and its documentation for 
educational purpose is hereby granted without fee, provided that the above copyright 
notice and this permission notice appear in all copies of this software and that you do 
not sell the software.
  
THE SOFTWARE IS PROVIDED "AS IS" AND WITHOUT WARRANTY OF ANY KIND,EXPRESS, IMPLIED OR 
OTHERWISE.

The CUDA Kernel for Applying BFS on a loaded Graph. Created By Pawan Harish
**********************************************************************************/
#ifndef _KERNEL_H_
#define _KERNEL_H_

__global__ void
Kernel( Node* g_graph_nodes, int* g_graph_edges, bool* g_graph_mask, bool* g_updating_graph_mask, bool *g_graph_visited, int* g_cost, int no_of_nodes) 
{
	int tid = blockIdx.x*MAX_THREADS_PER_BLOCK + threadIdx.x;
	
	// printf("Kernel#1 - Thread Id = %d - Before Check and Loop\n", tid);
	// printf("Kernel#1 - Thread Id = %d - Graph Mask (g_graph_mask[tid]) = %d\n", tid, g_graph_mask[tid]);
	// printf("Kernel#1 - Thread Id = %d - Number of Nodes (no_of_nodes) = %d\n", tid, no_of_nodes);
	
	if( tid<no_of_nodes && g_graph_mask[tid])
	{
		g_graph_mask[tid]=false;

		// printf("Kernel#1 - Thread Id = %d - After Check\n", tid);
		// printf("Kernel#1 - Thread Id = %d - Start (g_graph_nodes[tid].starting) = %d\n", tid, g_graph_nodes[tid].starting);
		// printf("Kernel#1 - Thread Id = %d - Count (g_graph_nodes[tid].no_of_edges + g_graph_nodes[tid].starting) = %d\n",
		//  tid, (g_graph_nodes[tid].no_of_edges + g_graph_nodes[tid].starting));
		// printf("Kernel#1 - Thread Id = %d - Number of Edges (g_graph_nodes[tid].no_of_edges) = %d\n",
		//  tid, (g_graph_nodes[tid].no_of_edges));
		
		for(int i=g_graph_nodes[tid].starting; i<(g_graph_nodes[tid].no_of_edges + g_graph_nodes[tid].starting); i++)
		{
			int id = g_graph_edges[i];

			// printf("Kernel#1 - Thread Id = %d - Enter Loop\n", tid);
			// printf("Kernel#1 - Thread Id = %d - Edge Id (g_graph_edges[i]) = %d\n", tid, g_graph_edges[i]);
			// printf("Kernel#1 - Thread Id = %d - Visited Flag (g_graph_visited[id]) = %d\n", tid, g_graph_visited[id]);

			if(!g_graph_visited[id])
			{
				g_cost[id]=g_cost[tid]+1;
				g_updating_graph_mask[id]=true;

				// printf("Kernel#1 - Thread Id = %d - Cost (g_cost[id]) = %d\n", tid, g_cost[id]);
			}
		}
	}
}

#endif 
