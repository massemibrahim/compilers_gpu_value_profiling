#include "hip/hip_runtime.h"
/*********************************************************************************
Implementing Breadth first search on CUDA using algorithm given in HiPC'07
  paper "Accelerating Large Graph Algorithms on the GPU using CUDA"

Copyright (c) 2008 International Institute of Information Technology - Hyderabad. 
All rights reserved.
  
Permission to use, copy, modify and distribute this software and its documentation for 
educational purpose is hereby granted without fee, provided that the above copyright 
notice and this permission notice appear in all copies of this software and that you do 
not sell the software.
  
THE SOFTWARE IS PROVIDED "AS IS" AND WITHOUT WARRANTY OF ANY KIND,EXPRESS, IMPLIED OR 
OTHERWISE.

The CUDA Kernel for Applying BFS on a loaded Graph. Created By Pawan Harish
**********************************************************************************/
#ifndef _KERNEL2_H_
#define _KERNEL2_H_

__global__ void
Kernel2( bool* g_graph_mask, bool *g_updating_graph_mask, bool* g_graph_visited, bool *g_over, int no_of_nodes)
{
	int tid = blockIdx.x*MAX_THREADS_PER_BLOCK + threadIdx.x;
	
	printf("Kernel#2 - Thread Id = %d - Before Check\n", tid);
	printf("Kernel#2 - Thread Id = %d - Number of Nodes (no_of_nodes) = %d\n", tid, no_of_nodes);
	printf("Kernel#2 - Thread Id = %d - Update Graph Mask (g_updating_graph_mask[tid]) = %d\n", tid, g_updating_graph_mask[tid]);

	if( tid<no_of_nodes && g_updating_graph_mask[tid])
	{
		g_graph_mask[tid]=true;
		g_graph_visited[tid]=true;
		*g_over=true;
		g_updating_graph_mask[tid]=false;

		printf("Kernel#2 - Thread Id = %d - After Check\n", tid);
		printf("Kernel#2 - Thread Id = %d - Update Graph Mask (g_updating_graph_mask[tid]) = %d\n", tid, g_updating_graph_mask[tid]);
		printf("Kernel#2 - Thread Id = %d - Visited Flag (g_graph_visited[tid]) = %d\n", tid, g_graph_visited[tid]);
		printf("Kernel#2 - Thread Id = %d - Over Flag (g_over) = %d\n", tid, g_over);
		printf("Kernel#2 - Thread Id = %d - Graph Mask (g_graph_mask[tid]) = %d\n", tid, g_graph_mask[tid]);
	}
}

#endif

