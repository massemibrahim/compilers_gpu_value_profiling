#include "hip/hip_runtime.h"


#ifndef _BACKPROP_CUDA_KERNEL_H_
#define _BACKPROP_CUDA_KERNEL_H_

#include <stdio.h>
#include "backprop.h"
#include "math.h"
#include "hip/hip_runtime.h"


__global__ void
bpnn_layerforward_CUDA(float *input_cuda,
	                   float *output_hidden_cuda,
					   float *input_hidden_cuda,
					   float *hidden_partial_sum,
					   int in,
					   int hid) 
{
   int by = blockIdx.y;
   int tx = threadIdx.x;
   int ty = threadIdx.y;
   // printf("Kernel#1 - Thread Id = %d/%d/%d\n", by, tx, ty);

   int index =  ( hid + 1 ) * HEIGHT * by + ( hid + 1 ) * ty + tx + 1 + ( hid + 1 ) ;  
   printf("Kernel#1 - Thread Id = %d/%d/%d - Index (index) = %d\n", by, tx, ty, index);

   int index_in = HEIGHT * by + ty + 1;
   printf("Kernel#1 - Thread Id = %d/%d/%d - Index In (index_in) = %d\n", by, tx, ty, index_in);

   __shared__ float input_node[HEIGHT];
   __shared__ float weight_matrix[HEIGHT][WIDTH];


   if ( tx == 0 )
   input_node[ty] = input_cuda[index_in];
   
   printf("Kernel#1 - Thread Id = %d/%d/%d - Input CUDA (input_cuda[index_in]) = %d\n", by, tx, ty, input_cuda[index_in]);
   printf("Kernel#1 - Thread Id = %d/%d/%d - Input Node (input_node[ty]) = %d\n", by, tx, ty, input_node[ty]);

   __syncthreads();

   weight_matrix[ty][tx] = input_hidden_cuda[index];

   printf("Kernel#1 - Thread Id = %d/%d/%d - Input Hidden CUDA (input_hidden_cuda[index]) = %d\n", by, tx, ty, input_hidden_cuda[index]);
   printf("Kernel#1 - Thread Id = %d/%d/%d - Weight Matrix (First) (weight_matrix[ty][tx]) = %d\n", by, tx, ty, weight_matrix[ty][tx]);   

   __syncthreads();
   
   weight_matrix[ty][tx] = weight_matrix[ty][tx] * input_node[ty];

   printf("Kernel#1 - Thread Id = %d/%d/%d - Weight Matrix (Second) (weight_matrix[ty][tx]) = %d\n", by, tx, ty, weight_matrix[ty][tx]);   

   __syncthreads();   
   
   for ( int i = 1 ; i <= __log2f(HEIGHT) ; i++){
 
	   int power_two = __powf(2, i);

	   if( ty % power_two == 0 )
	   weight_matrix[ty][tx] = weight_matrix[ty][tx] + weight_matrix[ty + power_two/2][tx];

      printf("Kernel#1 - Thread Id = %d/%d/%d - Weight Matrix (Third) (weight_matrix[ty][tx]) = %d\n", by, tx, ty, weight_matrix[ty][tx]);   

	   __syncthreads();

   }
   
   //__syncthreads();

   input_hidden_cuda[index] = weight_matrix[ty][tx];
   
/*
   for ( unsigned int i = 2 ; i <= HEIGHT ; i *= 2){
 
	   unsigned int power_two = i - 1;

	   if( (ty & power_two) == 0 ) {
		weight_matrix[ty][tx] = weight_matrix[ty][tx] + weight_matrix[ty + power_two/2][tx];
	   }

   }
   */

   __syncthreads();

   if ( tx == 0 ) {
	   hidden_partial_sum[by * hid + ty] = weight_matrix[tx][ty];

      printf("Kernel#1 - Thread Id = %d/%d/%d - Hidden Partial Sum (hidden_partial_sum[by * hid + ty]) = %d\n", by, tx, ty, hidden_partial_sum[by * hid + ty]);   
   }

}


__global__ void bpnn_adjust_weights_cuda(float * delta,   
										 int hid,         
										 float * ly,      
										 int in,          
										 float * w,       
										 float * oldw)  									
{
  
  
   int by = blockIdx.y;

   int tx = threadIdx.x;
   int ty = threadIdx.y;
	
   int index =  ( hid + 1 ) * HEIGHT * by + ( hid + 1 ) * ty + tx + 1 + ( hid + 1 ) ;  
   int index_y = HEIGHT * by + ty + 1;
   int index_x = tx + 1;
   //eta = 0.3;
   //momentum = 0.3;

   w[index] += ((ETA * delta[index_x] * ly[index_y]) + (MOMENTUM * oldw[index]));
   oldw[index] = ((ETA * delta[index_x] * ly[index_y]) + (MOMENTUM * oldw[index]));


   __syncthreads();

   if (ty == 0 && by ==0){
   w[index_x] += ((ETA * delta[index_x]) + (MOMENTUM * oldw[index_x]));
   oldw[index_x] = ((ETA * delta[index_x]) + (MOMENTUM * oldw[index_x]));
   }


}
#endif 
